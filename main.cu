#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdarg.h>
#include <hip/hip_runtime.h>

#define GPU __global__

#define GPU_DATA(TYPE, NAME, VALUE)   \
    TYPE* NAME;                        \
    NAME = (TYPE*)malloc(sizeof(TYPE)); \
    *NAME = (TYPE)VALUE;                 \
    TYPE* gpu_##NAME;                     \
    hipMalloc((void**)&gpu_##NAME, sizeof(TYPE));

#define GPU_DATA_N(TYPE, NAME, SIZE, VALUE)                    \
    TYPE* NAME;                                                 \
    NAME = (TYPE*)malloc(sizeof(TYPE) * SIZE));                  \
    { for (int i = 0; i < SIZE; ++i) { NAME[i] = (TYPE)VALUE; } } \
    TYPE* gpu_##NAME;                                              \
    hipMalloc((void**)&gpu_##NAME, sizeof(TYPE) * SIZE);           \
    
#define GPU_SEND(TYPE, NAME)           hipMemcpy(gpu_##NAME, NAME, sizeof(TYPE), hipMemcpyHostToDevice);
#define GPU_SEND_N(TYPE, NAME, SIZE)   hipMemcpy(gpu_##NAME, NAME, sizeof(TYPE) * SIZE, hipMemcpyHostToDevice);

#define GPU_SYNC(TYPE, NAME)           hipMemcpy(NAME, gpu_##NAME, sizeof(TYPE), hipMemcpyDeviceToHost);
#define GPU_SYNC_N(TYPE, NAME, SIZE)   hipMemcpy(NAME, gpu_##NAME, sizeof(TYPE) * SIZE, hipMemcpyDeviceToHost);

#define GPU_CALL(METHOD, ...)                    METHOD<<<1, 1>>>(__VA_ARGS__)
#define GPU_CALL_G_B(METHOD, GRID, BLOCK, ...)   METHOD<GRID, BLOCK>(__VA_ARGS__)

#define GPU_FREE(NAME) free(NAME); hipFree(gpu_##NAME);

GPU void cuda(float* a, float* b, float* c)
{
    printf("[CUDA]: Begin: %f + %f\n", *a, *b);
    //*c = *a + *b;
    asm("add.f32 %0,%1,%2;" : "=f"(*c) : "f"(*a), "f"(*b));
    printf("[CUDA]: End: %f\n", *c);
}

void delay(clock_t i)
{
    clock_t t1 = clock();
    clock_t t2 = 0;
    while (t2 < t1 + i) { t2 = clock(); }
}

int main()
{
    GPU_DATA(float, a, 1);
    GPU_DATA(float, b, 2); 
    GPU_DATA(float, c, 0);
    GPU_SEND(float, a);
    GPU_SEND(float, b);
    GPU_CALL(cuda, gpu_a, gpu_b, gpu_c);
    while (*c == 0.0f)
    {
        GPU_SYNC(float, c);
        printf("[INFO]: PTX-TEST: %f + %f = %f\n", *a, *b, *c);
        delay(1000);
    }
    GPU_FREE(a);
    GPU_FREE(b);
    GPU_FREE(c);
    return 0;
}

#define GPU_SLI(TYPE, SRC, DST)           hipMemcpy(DST, SRC, sizeof(TYPE), hipMemcpyDeviceToDevice);
#define GPU_SLI_N(TYPE, SRC, DST, SIZE)   hipMemcpy(DST, SRC, sizeof(TYPE) * SIZE, hipMemcpyDeviceToDevice);

#define CPU_SLI(TYPE, SRC, DST)           hipMemcpy(DST, SRC, sizeof(TYPE), hipMemcpyHostToHost);
#define CPU_SLI_N(TYPE, SRC, DST, SIZE)   hipMemcpy(DST, SRC, sizeof(TYPE) * SIZE, hipMemcpyHostToHost);
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    printf("[INFO]: %s\n", "TEST");
    double a = acos(1);
    float A = 1;
    float B = 2;
    float C = 0;
    int K = 1;
    int N = 10;
    VecAdd<<<K, N>>>(&A, &B, &C);
    return 0;
}